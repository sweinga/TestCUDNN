#include "hip/hip_runtime.h"
#include <iomanip>
#include <iostream>
#include <cstdlib>
#include <memory>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipDNN.h>

#include "utils.h"

const int x_w = 5;
const int x_h = 5;
const int x_c = 1;
const int x_n = 1;

const int w_w = 1;
const int w_h = 2;
const int w_c = 1;
const int w_k = 10;

const int pad_w = 0;
const int pad_h = 0;
const int str_w = 1;
const int str_h = 1;
const int dil_w = 1;
const int dil_h = 1;

const int x_bias = 1;
const int w_bias = 1;

__global__ void dev_const(float *px, float k) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = k;
}

__global__ void dev_iota(float *px, float bias) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = tid * 2 + bias;
}

__global__ void dev_iota2(float *px) {
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;
  px[tid] = (tid + 2) / 2;
}

int main() {
  ::hipdnnHandle_t cudnn;
  CUDNN_CALL(::hipdnnCreate(&cudnn));

  // input
  ::hipdnnTensorDescriptor_t x_desc;
  CUDNN_CALL(::hipdnnCreateTensorDescriptor(&x_desc));
  CUDNN_CALL(::hipdnnSetTensor4dDescriptor(
        x_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, x_n, x_c, x_h, x_w));

  // filter
  ::hipdnnFilterDescriptor_t w_desc;
  CUDNN_CALL(::hipdnnCreateFilterDescriptor(&w_desc));
  CUDNN_CALL(::hipdnnSetFilter4dDescriptor(
        w_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, w_k, w_c, w_h, w_w));

  // convolution
  ::hipdnnConvolutionDescriptor_t conv_desc;
  CUDNN_CALL(::hipdnnCreateConvolutionDescriptor(&conv_desc));
#if CUDNN_MAJOR >= 6
  CUDNN_CALL(::hipdnnSetConvolution2dDescriptor(
        conv_desc,
        pad_h, pad_w, str_h, str_w, dil_h, dil_w,
        HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT));
#else
  CUDNN_CALL(::hipdnnSetConvolution2dDescriptor(
        conv_desc,
        pad_h, pad_w, str_h, str_w, dil_h, dil_w,
        HIPDNN_CONVOLUTION));
#endif  // CUDNN_MAJOR

  // output
  int y_n, y_c, y_h, y_w;
  CUDNN_CALL(::hipdnnGetConvolution2dForwardOutputDim(
        conv_desc, x_desc, w_desc, &y_n, &y_c, &y_h, &y_w));

  ::hipdnnTensorDescriptor_t y_desc;
  CUDNN_CALL(::hipdnnCreateTensorDescriptor(&y_desc));
  CUDNN_CALL(::hipdnnSetTensor4dDescriptor(
        y_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, y_n, y_c, y_h, y_w));

  // algorithms
  ::hipdnnConvolutionFwdAlgo_t fwd_algo;
  CUDNN_CALL(::hipdnnGetConvolutionForwardAlgorithm(
        cudnn,
        x_desc, w_desc, conv_desc, y_desc,
        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &fwd_algo));

  // workspaces
  size_t fwd_ws_size;
  CUDNN_CALL(::hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn, x_desc, w_desc, conv_desc, y_desc, fwd_algo, &fwd_ws_size));

  // memories
  //auto x_data = ::allocate<float>(x_n * x_c * x_h * x_w * sizeof(float));
  float *x_data;
  ::hipMalloc( (void**) &x_data, x_n * x_c * x_h * x_w * sizeof(float) );
  auto w_data = ::allocate<float>(w_k * w_c * w_h * w_w * sizeof(float));
  auto y_data = ::allocate<float>(y_n * y_c * y_h * y_w * sizeof(float));
  auto gy_data = ::allocate<float>(y_n * y_c * y_h * y_w * sizeof(float));
  auto gx_data = ::allocate<float>(x_n * x_c * x_h * x_w * sizeof(float));
  auto gw_data = ::allocate<float>(w_k * w_c * w_h * w_w * sizeof(float));
  auto fwd_ws_data = ::allocate(fwd_ws_size);

  // initialize inputs
  dev_iota<<<x_w * x_h, x_n * x_c>>>(x_data, x_bias);
  //dev_iota<<<w_w * w_h, w_k * w_c>>>(w_data.get(), w_bias);
  dev_iota2<<<w_w * w_h, w_k * w_c>>>(w_data.get());
  dev_const<<<y_w * y_h, y_n * y_c>>>(gy_data.get(), 1);
  dev_const<<<x_w * x_h, x_n * x_c>>>(gx_data.get(), 0);
  dev_const<<<w_w * w_h, w_k * w_c>>>(gw_data.get(), 0);

  // perform forward operation
  float fwd_alpha = 1.f;
  float fwd_beta = 0.f;
  CUDNN_CALL(::hipdnnConvolutionForward(
        cudnn,
        &fwd_alpha, x_desc, x_data, w_desc, w_data.get(),
        conv_desc, fwd_algo, fwd_ws_data.get(), fwd_ws_size,
        &fwd_beta, y_desc, y_data.get()));

  // results
  std::cout << "x_w: " << x_w << std::endl;
  std::cout << "x_h: " << x_h << std::endl;
  std::cout << "x_c: " << x_c << std::endl;
  std::cout << "x_n: " << x_n << std::endl;
  std::cout << std::endl;
  std::cout << "w_w: " << w_w << std::endl;
  std::cout << "w_h: " << w_h << std::endl;
  std::cout << "w_c: " << w_c << std::endl;
  std::cout << "w_k: " << w_k << std::endl;
  std::cout << std::endl;
  std::cout << "pad_w: " << pad_w << std::endl;
  std::cout << "pad_h: " << pad_h << std::endl;
  std::cout << "str_w: " << str_w << std::endl;
  std::cout << "str_h: " << str_h << std::endl;
  std::cout << "dil_w: " << dil_w << std::endl;
  std::cout << "dil_h: " << dil_h << std::endl;
  std::cout << std::endl;
  std::cout << "y_w: " << y_w << std::endl;
  std::cout << "y_h: " << y_h << std::endl;
  std::cout << "y_c: " << y_c << std::endl;
  std::cout << "y_n: " << y_n << std::endl;
  std::cout << std::endl;

  std::cout << "Algorithm (fwd): " << fwd_algo << std::endl;
  std::cout << "Workspace size (fwd): " << fwd_ws_size << std::endl;
  std::cout << std::endl;

  std::cout << "x_data:" << std::endl;
  print(&x_data[0], x_n, x_c, x_h, x_w);
  std::cout << "w_data:" << std::endl;
  print(w_data.get(), w_k, w_c, w_h, w_w);
  std::cout << "y_data:" << std::endl;
  print(y_data.get(), y_n, y_c, y_h, y_w);
  std::cout << "gy_data:" << std::endl;
  print(gy_data.get(), y_n, y_c, y_h, y_w);
  std::cout << "gx_data:" << std::endl;
  print(gx_data.get(), x_n, x_c, x_h, x_w);
  std::cout << "gw_data:" << std::endl;
  print(gw_data.get(), w_k, w_c, w_h, w_w);

  // finalizing
  CUDNN_CALL(::hipdnnDestroyTensorDescriptor(y_desc));
  CUDNN_CALL(::hipdnnDestroyConvolutionDescriptor(conv_desc));
  CUDNN_CALL(::hipdnnDestroyFilterDescriptor(w_desc));
  CUDNN_CALL(::hipdnnDestroyTensorDescriptor(x_desc));
  CUDNN_CALL(::hipdnnDestroy(cudnn));
  return 0;
}
